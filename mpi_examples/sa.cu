#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <random>
#include <math.h>
#include <unistd.h>
#define PI 3.1415926535
#define RANGE 600

__host__ __device__ double ackley(int DIMENSION, double *point, int offset);//RANGE = 1
__host__ __device__ double griewangk(int DIMENSION, double *point, int offset);//RANGE = 600
__host__ __device__ double schwefel(int DIMENSION, double *point, int offset);//RANGE = 500
__host__ __device__ double dejong(int DIMENSION, double *point, int offset);//RANGE = 5.12
__host__ __device__ double rastrigin(int DIMENSION, double *point, int offset);//RANGE = 5.12

//Replace with a predefined function and set RANGE accordingly 
__host__ __device__ double fitness(int DIMENSION, double *point, int offset){
  return griewangk(DIMENSION, point, offset);
}

__host__ __device__ double ackley(int DIMENSION, double *point, int offset){
  double sum_result1 = 0;
  double sum_result2 = 0;
  for(int i=0; i<DIMENSION; i++){
    sum_result1 += point[offset+i]*point[offset+i];
    sum_result2 += cos(2*PI*point[offset+i]);
  }
  return -20*exp(-0.2*sqrt(sum_result1/DIMENSION))-exp(sum_result2/DIMENSION)+20+exp(1.);
}

__host__ __device__ double griewangk(int DIMENSION, double *point, int offset){
  double sum_result = 0;
  double product_result = 1;
  for(int i=0; i<DIMENSION; i++){
    sum_result += point[offset+i]/4000*point[offset+i];
    product_result *= cos(point[offset+i]/sqrt((double)i+1));
  }
  return sum_result-product_result+1;
}

__host__ __device__ double schwefel(int DIMENSION, double *point, int offset){
  double result = 0;
  for(int i=0; i<DIMENSION; i++){
    result -= point[offset+i]*sin(sqrt(abs(point[offset+i])));
  }
  return result+DIMENSION*418.9829;//shifted to obtain 0 minimum
}

__host__ __device__ double dejong(int DIMENSION, double *point, int offset){
  double result = 0;
  for(int i=0; i<DIMENSION; i++){
    result += point[offset+i]*point[offset+i];
  }
  return result;
}

__host__ __device__ double rastrigin(int DIMENSION, double *point, int offset){
  double result = DIMENSION*10;
  for(int i=0; i<DIMENSION; i++){
    result += point[offset+i]*point[offset+i]-10*cos(2*PI*point[offset+i]);
  }
  return result;
}

__global__ void curand_init_kernel(hiprandState *state, int seed){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, tid, 0, &state[tid]);
}

__global__ void sa_kernel(int INSTANCES, int DIMENSION, int GPU_ITERATIONS, hiprandState *state, double *decay, double *point, double *neighbor, double *temperature) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < INSTANCES) {
    for(int i=0; i<GPU_ITERATIONS; i++){
      int axis = trunc(hiprand_uniform_double(&state[tid])*DIMENSION);
      neighbor[tid*DIMENSION+axis] = (hiprand_uniform_double(&state[tid])*2-1)*RANGE;
      double point_fitness=fitness(DIMENSION, point, DIMENSION*tid);
      double neighbor_fitness=fitness(DIMENSION, neighbor, DIMENSION*tid);
      double change_prob = 2; //ensure change
       if(neighbor_fitness>point_fitness) change_prob = exp((point_fitness-neighbor_fitness)/temperature[tid]);
      if(change_prob > hiprand_uniform_double(&state[tid]))
        point[tid*DIMENSION+axis] = neighbor[tid*DIMENSION+axis];
      else
        neighbor[tid*DIMENSION+axis] = point[tid*DIMENSION+axis];
      temperature[tid]*=decay[tid];
    }
  }
}

int main(int argc, char **argv) {
  if(argc<5) {
    printf("Required arguments:\nINSTANCES - less than 1024\nDIMENSION - size of a problem\nITERATIONS - number of times the data will be synchronized with GPUs\nGPU_ITERATIONS - number of iterations per core in a single synchronization step\n");
    return 0;
  }
  const int INSTANCES = atoi(argv[1]);
  const int DIMENSION = atoi(argv[2]);
  const int ITERATIONS = atoi(argv[3]);
  const int GPU_ITERATIONS = atoi(argv[4]);
  time_t time_measure = time(0);
  
  const int RANDOM_SEED = time(0)*getpid();

  hiprandState* dev_state;
  hipMalloc((void**) &dev_state, INSTANCES * sizeof(hiprandState));
  curand_init_kernel<<<1,INSTANCES>>>(dev_state, RANDOM_SEED);
  
  std::default_random_engine generator(static_cast<long unsigned int>(RANDOM_SEED));
  std::uniform_real_distribution<double> decay_distribution(0.9,1.0);
  std::uniform_real_distribution<double> point_distribution(-RANGE,RANGE);
  
  double *decay = new double[INSTANCES];
  for(int i=0; i<INSTANCES; i++){
    decay_distribution.reset();
    decay[i] = decay_distribution(generator);
    //printf("DECAY FACTOR %d: %f\n",i,decay[i]);
  }
  double *dev_decay;
  hipMalloc((void**)&dev_decay, INSTANCES * sizeof(double));
  hipMemcpy(dev_decay, decay, INSTANCES * sizeof(double), hipMemcpyHostToDevice);
  
  double *point = new double[INSTANCES*DIMENSION];
  double *neighbor = new double[INSTANCES*DIMENSION];
  for(int i=0; i<INSTANCES; i++){
    for(int d=0; d<DIMENSION; d++){
      point_distribution.reset();
      point[i*DIMENSION+d]=point_distribution(generator);
      neighbor[i*DIMENSION+d]=point[i*DIMENSION+d];
    }
  }
  double *dev_point, *dev_neighbor;
  hipMalloc((void**)&dev_point, INSTANCES * DIMENSION * sizeof(double));
  hipMemcpy(dev_point, point, INSTANCES * DIMENSION * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&dev_neighbor, INSTANCES * DIMENSION * sizeof(double));
  hipMemcpy(dev_neighbor, neighbor, INSTANCES * DIMENSION * sizeof(double), hipMemcpyHostToDevice);

  double *temperature = new double[INSTANCES];
  for(int i=0; i<INSTANCES; i++){
    temperature[i] = 100.0;
  }
  double *dev_temperature;
  hipMalloc((void**)&dev_temperature, INSTANCES * sizeof(double));
  hipMemcpy(dev_temperature, temperature, INSTANCES * sizeof(double), hipMemcpyHostToDevice);
  
  //######################## KERNEL ########################
  for(int i=0; i<ITERATIONS; i++){
    sa_kernel<<<1,INSTANCES>>>(INSTANCES, DIMENSION, GPU_ITERATIONS, dev_state, dev_decay, dev_point, dev_neighbor, dev_temperature);
    hipDeviceSynchronize();
    hipMemcpy(point, dev_point, INSTANCES * DIMENSION * sizeof(double), hipMemcpyDeviceToHost);
    //Pick the minimal point
    int minimum_index = 0;
    double minimum_value = fitness(DIMENSION,point,0);
    for(int k=1; k<INSTANCES; k++){
      double minimum_value_contestant = fitness(DIMENSION,point,DIMENSION*k);
      if(minimum_value_contestant<minimum_value){
        minimum_value = minimum_value_contestant;
        minimum_index = k;
      }
    }
    //Copy minimal point
    for(int k=0; k<INSTANCES; k++){
      for(int t=0; t<DIMENSION; t++){
        point[DIMENSION*k+t] = point[DIMENSION*minimum_index+t];
      }
    }
    hipMemcpy(dev_point, point, INSTANCES * DIMENSION * sizeof(double), hipMemcpyHostToDevice);
  }
  
  hipMemcpy(point, dev_point, INSTANCES * DIMENSION * sizeof(double), hipMemcpyDeviceToHost);
  printf("%d\t%.10f\n", time(0)-time_measure, fitness(DIMENSION,point,0));

  delete [] decay;
  delete [] neighbor;
  delete [] point;
  delete [] temperature;
  hipFree(dev_decay);
  hipFree(dev_neighbor);
  hipFree(dev_point);
  hipFree(dev_temperature);
  return 0;
}
