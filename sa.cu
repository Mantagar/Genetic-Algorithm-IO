#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <helper_timer.h>
#include <random>
#include <math.h>
#define PI 3.1415926535
__host__ __device__ double rastrigin(int DIMENSION, double *point, int offset){
  double result = DIMENSION*10;
  for(int i=0; i<DIMENSION; i++){
    result+=point[offset+i]*point[offset+i]-10*cos(2*PI*point[offset+i]);
  }
  return result;
}

__global__ void curand_init_kernel(hiprandState *state){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(0, tid, 0, &state[tid]);
}

__global__ void sa_kernel(int INSTANCES, int DIMENSION, hiprandState *state, double *decay, double *point, double *neighbor, double *temperature) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < INSTANCES) {
    int axis = trunc(hiprand_uniform_double(&state[tid])*DIMENSION);
    neighbor[tid*DIMENSION+axis] = (hiprand_uniform_double(&state[tid])-0.5)*10.24;
    double point_fitness=rastrigin(DIMENSION, point, DIMENSION*tid);
    double neighbor_fitness=rastrigin(DIMENSION, neighbor, DIMENSION*tid);
    double change_prob = 2; //ensure change
    if(neighbor_fitness>point_fitness) change_prob = exp((point_fitness-neighbor_fitness)/temperature[tid]);
    if(change_prob > hiprand_uniform_double(&state[tid]))
      point[tid*DIMENSION+axis] = neighbor[tid*DIMENSION+axis];
    else
      neighbor[tid*DIMENSION+axis] = point[tid*DIMENSION+axis];
    temperature[tid]*=decay[tid];
  }
}

int main(int argc, char **argv) {
  if(argc<4) {
    printf("Required arguments:\nINSTANCES - number of CUDA threads\nDIMENSION - size of a problem\nITERATIONS - number of iterations");
    return 0;
  }
  const int INSTANCES = atoi(argv[1]);
  const int DIMENSION = atoi(argv[2]);
  const int ITERATIONS = atoi(argv[3]);


  hiprandState* dev_state;
  hipMalloc((void**) &dev_state, INSTANCES * sizeof(hiprandState));
  curand_init_kernel<<<1,INSTANCES>>>(dev_state);

  std::default_random_engine generator(static_cast<long unsigned int>(time(0)));
  std::uniform_real_distribution<double> decay_distribution(0.9,1.0);
  std::uniform_real_distribution<double> point_distribution(-5.12,5.12);

  double *decay = new double[INSTANCES];
  for(int i=0; i<INSTANCES; i++){
    decay_distribution.reset();
    decay[i] = decay_distribution(generator);
    printf("DECAY FACTOR %d: %f\n",i,decay[i]);
  }
  double *dev_decay;
  hipMalloc((void**)&dev_decay, INSTANCES * sizeof(double));
  hipMemcpy(dev_decay, decay, INSTANCES * sizeof(double), hipMemcpyHostToDevice);

  double *point = new double[INSTANCES*DIMENSION];
  double *neighbor = new double[INSTANCES*DIMENSION];
  for(int i=0; i<INSTANCES; i++){
    for(int d=0; d<DIMENSION; d++){
      point_distribution.reset();
      point[i*DIMENSION+d]=point_distribution(generator);
      neighbor[i*DIMENSION+d]=point[i*DIMENSION+d];
    }
  }
  double *dev_point, *dev_neighbor;
  hipMalloc((void**)&dev_point, INSTANCES * DIMENSION * sizeof(double));
  hipMemcpy(dev_point, point, INSTANCES * DIMENSION * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&dev_neighbor, INSTANCES * DIMENSION * sizeof(double));
  hipMemcpy(dev_neighbor, neighbor, INSTANCES * DIMENSION * sizeof(double), hipMemcpyHostToDevice);
  for(int i=0; i<INSTANCES; i++){
    //for(int d=0; d<DIMENSION; d++) printf("%f\n",point[i*DIMENSION+d]);
    printf("RASTRIGIN %d: %f\n",i,rastrigin(DIMENSION,point,DIMENSION*i));
  }

  double *temperature = new double[INSTANCES];
  for(int i=0; i<INSTANCES; i++){
    temperature[i] = 100.0;
  }
  double *dev_temperature;
  hipMalloc((void**)&dev_temperature, INSTANCES * sizeof(double));
  hipMemcpy(dev_temperature, temperature, INSTANCES * sizeof(double), hipMemcpyHostToDevice);

  StopWatchInterface *timer=NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  //######################## KERNEL ########################
  for(int i=0; i<ITERATIONS; i++){
    sa_kernel<<<1,INSTANCES>>>(INSTANCES, DIMENSION, dev_state, dev_decay, dev_point, dev_neighbor, dev_temperature);
    hipDeviceSynchronize();
    hipMemcpy(point, dev_point, INSTANCES * DIMENSION * sizeof(double), hipMemcpyDeviceToHost);
    //Pick the minimal point
    int minimum_index = 0;
    int minimum_value = rastrigin(DIMENSION,point,0);
    for(int k=1; k<INSTANCES; k++){
      int minimum_value_contestant = rastrigin(DIMENSION,point,DIMENSION*k);
      if(minimum_value_contestant<minimum_value){
        minimum_value = minimum_value_contestant;
        minimum_index = k;
      }
    }
    //Copy minimal point
    for(int k=0; k<INSTANCES; k++){
      for(int t=0; t<DIMENSION; t++){
        point[DIMENSION*k+t] = point[DIMENSION*minimum_index+t];
      }
    }
    hipMemcpy(dev_point, point, INSTANCES * DIMENSION * sizeof(double), hipMemcpyHostToDevice);
    //printf("Iteration no %d, leading instance: %d\n",i,minimum_index);
  }

  sdkStopTimer(&timer);
  float time = sdkGetTimerValue(&timer);
  sdkDeleteTimer(&timer);

  printf("%f ms\n",time);

  hipMemcpy(point, dev_point, INSTANCES * DIMENSION * sizeof(double), hipMemcpyDeviceToHost);
  for(int i=0; i<INSTANCES; i++){
    //for(int d=0; d<DIMENSION; d++) printf("%f\n",point[i*DIMENSION+d]);
    printf("RASTRIGIN %d: %f\n",i,rastrigin(DIMENSION,point,DIMENSION*i));
  }

  delete [] decay;
  delete [] neighbor;
  delete [] point;
  delete [] temperature;
  hipFree(dev_decay);
  hipFree(dev_neighbor);
  hipFree(dev_point);
  hipFree(dev_temperature);
  return 0;
}
